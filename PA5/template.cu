#include "hip/hip_runtime.h"
#include <gputk.h>

#define gpuTKCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      gpuTKLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define Mask_width 5
#define Mask_radius Mask_width / 2
#define TILE_WIDTH 16
#define w (TILE_WIDTH + Mask_width - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))

//@@ INSERT CODE HERE
__global__ void convolution(float *inputImage, float *mask, float *outputImage,
                             int channels, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int channel = threadIdx.z;

    if (row < height && col < width) {
        float sum = 0.0f;
        for (int i = -Mask_radius; i <= Mask_radius; ++i) {
            for (int j = -Mask_radius; j <= Mask_radius; ++j) {
                int curRow = row + i;
                int curCol = col + j;
                if (curRow >= 0 && curRow < height && curCol >= 0 && curCol < width) {
                    float pixel = inputImage[(curRow * width + curCol) * channels + channel];
                    float maskValue = mask[(i + Mask_radius) * Mask_width + j + Mask_radius];
                    sum += pixel * maskValue;
                }
            }
        }
        outputImage[(row * width + col) * channels + channel] = clamp(sum);
    }
}

int main(int argc, char *argv[]) {
  gpuTKArg_t arg;
  int maskRows;
  int maskColumns;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  char *inputMaskFile;
  gpuTKImage_t inputImage;
  gpuTKImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *hostMaskData;
  float *deviceInputImageData;
  float *deviceOutputImageData;
  float *deviceMaskData;

  arg = gpuTKArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = gpuTKArg_getInputFile(arg, 0);
  inputMaskFile  = gpuTKArg_getInputFile(arg, 1);

  inputImage   = gpuTKImport(inputImageFile);
  hostMaskData = (float *)gpuTKImport(inputMaskFile, &maskRows, &maskColumns);

  assert(maskRows == 5);    /* mask height is fixed to 5 in this mp */
  assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

  imageWidth    = gpuTKImage_getWidth(inputImage);
  imageHeight   = gpuTKImage_getHeight(inputImage);
  imageChannels = gpuTKImage_getChannels(inputImage);

  outputImage = gpuTKImage_new(imageWidth, imageHeight, imageChannels);

  hostInputImageData  = gpuTKImage_getData(inputImage);
  hostOutputImageData = gpuTKImage_getData(outputImage);

  gpuTKTime_start(GPU, "Doing GPU Computation (memory + compute)");

  gpuTKTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void**)&deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void**)&deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void**)&deviceMaskData, Mask_width * Mask_width * sizeof(float));
  gpuTKTime_stop(GPU, "Doing GPU memory allocation");

  gpuTKTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceMaskData, hostMaskData, Mask_width * Mask_width * sizeof(float), hipMemcpyHostToDevice);
  gpuTKTime_stop(Copy, "Copying data to the GPU");

  gpuTKTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 3);
  dim3 dimGrid((imageWidth) / TILE_WIDTH + 1, (imageHeight) / TILE_WIDTH + 1);


  convolution<<<dimGrid, dimBlock>>>(deviceInputImageData, deviceMaskData,
                                     deviceOutputImageData, imageChannels,
                                     imageWidth, imageHeight);
  gpuTKTime_stop(Compute, "Doing the computation on the GPU");

  gpuTKTime_start(Copy, "Copying data from the GPU");
  //@@ INSERT CODE HERE
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyDeviceToHost);
  gpuTKTime_stop(Copy, "Copying data from the GPU");

  gpuTKTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  gpuTKSolution(arg, outputImage);

  //@@ Insert code here
  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);
  hipFree(deviceMaskData);

  free(hostMaskData);
  gpuTKImage_delete(outputImage);
  gpuTKImage_delete(inputImage);

  return 0;
}
