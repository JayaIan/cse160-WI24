#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

# define TILE_WIDTH 32

// __constant__ float masks[49];

__global__ void conv_forward_kernel(float *y, const float *x, const float *k, const int B, const int M, const int C, const int H, const int W, const int K)
{
// An example use of these macros:
// float a = y4d(0,0,0,0)
// y4d(0,0,0,0) = a

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.
    We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    */


    // __shared__ float sharedWeights[M][C][K][K];
    // __shared__ float subTileN[TILE_WIDTH][TILE_WIDTH];

    int W_grid = ceil(W_out*1.0/TILE_WIDTH); 	// number of horizontal tiles per output map

    int b = blockIdx.z;
    int m = blockIdx.x;
    int h = (blockIdx.y / W_grid) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.y % W_grid) * TILE_WIDTH + threadIdx.x;
    
    float acc = 0.;
    if (h < H_out && w < W_out) {
        for (int c = 0;  c < C; c++) {		// sum over all input channels
            for (int p = 0; p < K; p++) {		// loop over KxK  filter
                for (int q = 0; q < K; q++)  {
                    acc += x4d(b, c, h+p, w+q) * k4d(m, c, p, q);
                }
            }
        }
        y4d(b, m, h, w) = acc;
    } else {
        return;
    }


#undef y4d
#undef x4d
#undef k4d
}


	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_y, const float *host_x, const float *host_k, float **device_y_ptr, float **device_x_ptr, float **device_k_ptr, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU
    printf("allocating memory");
    hipMalloc((void**)device_y_ptr, B * M * (H - K + 1) * (W - K + 1) * sizeof(float));
    hipMalloc((void**)device_x_ptr, B * C * H * W * sizeof(float));
    hipMalloc((void**)device_k_ptr, M * C * K * K * sizeof(float));

    hipMemcpy(*device_x_ptr, host_x, B * C * H * W * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(*device_y_ptr, host_y, B * M * (H - K + 1) * (W - K + 1) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_k_ptr, host_k, M * C * K * K * sizeof(float), hipMemcpyHostToDevice);   
    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_y, const float *device_x, const float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    int H_out = H - K + 1;
    int W_out = W - K + 1;
    int W_grid = ceil(W_out*1.0/TILE_WIDTH); 	// number of horizontal tiles per output map
    int H_grid = ceil(H_out*1.0/TILE_WIDTH); 	// number of vertical tiles per output map
    printf("w_grid %d", W_grid);
    printf("h_grid %d", H_grid);
    int Y = H_grid * W_grid;		// Number of blocks in the Y dimension
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 gridDim(M, Y, B);
    conv_forward_kernel<<< gridDim, blockDim>>>(device_y, device_x, device_k, B, M, C, H, W, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_y, float *device_y, float *device_x, float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Copy the output back to host
    hipMemcpy(host_y, device_y, B * M * (H - K + 1) * (W - K + 1) * sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_y);
    hipFree(device_x);
    hipFree(device_k);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
